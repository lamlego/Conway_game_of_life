
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sstream>
#include <chrono>	 // timing libraries
#include <vector>
#include <algorithm>
#include <omp.h>
using namespace std;

namespace gpu {
     //takes pointers
     void split(bool **world,bool **top, bool **bot,int m, int n){
        m = m+2;
        int count = m;
        bool *t = *top;
        bool *w = *world;
        bool *b = *bot;
        for(int i = 0; i< m*n; i++){
            t[count]  = w[i];
            //cout<<w[i];
            count++;
        }
        count = 0;
        //cout<<"top filled\n";
        for(int i = m; i< m*n; i++){
    
            b[count] = w[i];
            count++;
        }
    }
    __global__
        void next_turn(bool *top, bool *core, bool* bot, bool *result,int m, int n){
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            int living = 0;
            //printf("%d",i);
            bool current = core[i];
            //result[i] = current;
            //__syncthreads();
            if(i < m*n){
                //printf("h%d ",i);
                if(i%m != 0 &&(i+1)%m!=0){ //
                    //printf("i%d ",i);
                    living = top[i-1] + top[i] + top[i+1] +
                             core[i-1]+         core[i+1] +
                             bot[i-1] + bot[i] + bot[i+1];
                    //printf("%d living: %d\n",i,living);
                    if(living == 3 || (current&& living ==2)){
                        result[i] = true;
                    }else{
                        result[i] = false;
                    }
                }
                //printf("%d",result[i]);
            }
            
          
        }


    auto bench_mark(void(*func)(bool *,bool *,bool *,bool *,int,int),bool *world, int m, int n,int it){
        
            int size = (n*(m+2))*sizeof(bool);
            bool *top, *bot;
            bool *d_top, *d_core, *d_bot, *d_result;


            hipMalloc((void**)&d_top,size);
            hipMalloc((void **)&d_core,size);
            hipMalloc((void **)&d_bot,size);
            hipMalloc((void **)&d_result,size);
            //cudaMallocManaged((void**)&d_result,size);
            
            //top = (bool *)malloc(size);
            //bot = (bool *)malloc(size);
            top = new bool[size]();
            bot = new bool[size]();
            
            auto const start_time = std::chrono::steady_clock::now();

            for (int i=0; i< it; i++){
                //call cuda function here
                gpu::split(&world,&top,&bot,m,n);
                //cout<<"-----result-------\n";
                hipMemcpy(d_top, top, size, hipMemcpyHostToDevice);
                hipMemcpy(d_core, world, size, hipMemcpyHostToDevice);
                hipMemcpy(d_bot, bot, size, hipMemcpyHostToDevice);
                gpu::next_turn<<<ceil((m+2)*n/1024),1024>>>(d_top,d_core,d_bot,d_result,m+2,n);
                hipDeviceSynchronize();
                hipMemcpy(world, &d_result,size,hipMemcpyDeviceToHost);
                
            }

            auto const end_time = std::chrono::steady_clock::now();
            
            // for(int i = 0;i<n;i++){
            //     for(int j =0;j<(m+2);j++){
            //         cout<<world[i*(m+2)+j];
            //     }
            // cout<< endl;
            // }
            // free(top);free(bot);
            // cudaFree(d_core);cudaFree(d_top);cudaFree(d_bot);cudaFree(d_result);
            return(std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time ).count());
            //cout<< std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time ).count() << " micro seconds\n";
            }
        
        
       

        bool* create_world(int m, int n){
            m = m+2;
            bool * world;// grid to hold randomly generated world
            world = new bool[m*n]();
            srand(time(0));
            //assigning random values to world except for borders
            for(int i = 0; i < n ; i++){
                for(int j = 1; j< m-1; j++){
                    world[(i*m)+j] = rand()%2;
                }
            }
            return world;
        }



}
namespace cpu{
/**takes the number of living cell and the current bit
 * returns an boolean
 */
    bool dead_or_alive(int living, bool current){
        bool newcell = current;
        //std::cout<< current <<" ";
        if((current == 1 && (living == 3 || living == 2)) || (current == 0 && living == 3)){
            newcell = 1;
        }
        else {
        newcell = 0;
        }
        
        return newcell;
    }
    //creates a matrix of m by n and loading cells with 0|1 and surround it with a border of 0
    bool * create_world(int m, int n)
    {
        bool* world;
        //creating the array so that it is loaded with 0s to avoid if statments. 
        world = new bool[m*n]();
        
        srand(time(0));
        for(int i = 0; i < m ; i++){
            //only setting the inside box to randomized values of 1 or 0
            for(int j = 1; j< n-1; j++){
                world[i*m+j] = rand()%2;
            }
        }
        return world;
    }

    //takes a 2d int matrix and dimentions and print it
    void print_matrix(bool* world,int m, int n){
        for (int i = 0; i < m; i++){
                cout<<"| ";
                for(int j =0; j < n; j++){
                    cout<<world[i*m+j]<<" ";
                }
                cout<<"|\n";
            }
    }

    //return the time it take to run next turn for it number of iterations
    auto bench_mark(bool*(*func)(bool*,int,int),bool* world, int m, int n,int it){
    auto const start_time = std::chrono::steady_clock::now();
    for(int i = 0; i < it; i++){


        world = func(world,m,n);
    }
    auto const end_time = std::chrono::steady_clock::now();
    return(std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time ).count());
    //cout<< std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time ).count() << " micro seconds\n";
    }

    /**
    * takes a 2d int matrix,and the dimentions
    * goes through each cell
    * determine the next frame of the matrix 
    */
    bool * next_turn(bool* world,int m,int n)
    {
        bool* new_world;
        new_world= new bool[m*n]();
        //#pragma omp parallel for
        for(int i = 1;i < m-1 ;i++){ 
            for(int j = 0; j < n; j++){
                int living = world[(i-1)*m+(j-1)] + world[(i-1)*m+j] + world[(i-1)*m+(j+1)] +
                            world[i*m+(j+1)] + world[i*m+(j-1)] + 
                            world[(i+1)*m+(j-1)] + world[(i+1)*m+j] + world[(i+1)*m+(j+1)];
                new_world[i*m+j] = dead_or_alive(living, world[i*m+j]);
            }
        }
        //free(world); this is slower
        delete[] world;
        return new_world;
    }
}
int main(int argc, char *argv[]){
    if(argc <5 || argc> 6){
        cout<<"Usage: ./gol [width] [height] [iterations] [number of tests][number of threads](if not decleared use all)]";
        return 1;
    }
    //set size larger so we can add border of 0 surrounding the map
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int num_tests = atoi(argv[4]);
    // if(argc==6){
    //     omp_set_num_threads(atoi(argv[5]));
    // }
    //Checking data that was entered making sure it is INT
    if((m == 0) || (n == 0) || (iterations == 0) || (num_tests == 0)){
        cout<< "must enter INT values";
        return 0;
    }
    int time = 0;
    vector<bool*> test_cases;
    //create test cases into a vector
    for (int i = 0; i < num_tests; i++){
        test_cases.push_back(gpu::create_world(m,n));
    }
    // for(int i = 0;i<n;i++){
    //     for(int j =0;j<m+2;j++){
    //         cout<<test_cases[0][i*(m+2)+j];
    //     }
    // cout<< endl;
    // }
    // cout<<"------------\n";
    //run each case
    for(vector<bool*>::iterator i = test_cases.begin(); i != test_cases.end(); ++i){
        time +=gpu::bench_mark(gpu::next_turn,*i,m,n,iterations);
    }
    //print out the average
    cout<< "ran "<< num_tests << " random games of "<< n << " by "<< m << " for "<< iterations<< " iterations, average time is: "<< time/num_tests<<" us"<<endl;
    /*
    //uncomment this section to print out a iteration
    bool* world= create_world(m,n);
    for(int i=0;i<m*n;i++){
        world[i] =0;
    }
    //this creates stable pattern, turns into a 6x6 cross without center in 3rd turn
    world[(m/2)*m+(n/2)-2] =1;
    world[(m/2)*m+(n/2) -1] =1;
    world[(m/2)*m+(n/2)+1] =1;
    world[(m/2)*m+(n/2)+2] =1;
    world[(m/2-1)*m+(n/2)] =1;
    world[(m/2-2)*m+(n/2)] =1;
    world[(m/2-2)*m+(n/2)-2] =1;
    world[(m/2-2)*m+(n/2)+2] =1;
    world[(m/2+1)*m+(n/2)] =1;
    world[(m/2+2)*m+(n/2)] =1;
    world[(m/2+2)*m+(n/2-2)] =1;
    world[(m/2+2)*m+(n/2+2)] =1;
    cout << "our initial matrix\n";
    print_matrix(world,m,n);
    for(int iter = 0; iter < iterations; iter++){// print out each iteration of the matric
        cout<<"\n";
        cout<< iter+1 << "th iteration\n";
        world = next_turn(world,m,n);
        print_matrix(world,m,n);
    }
    */
    
    
    return 0;
}