
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sstream>
#include <chrono>	 // timing libraries
#include <vector>
#include <algorithm>
#include <omp.h>
using namespace std;
/**
*This is an attempt at Conway's game of life in cuda using 
*simt implentation. Right now it has multiple problems and errors 
*that leads to crash
*/

//this name space contain all the funtions that we use in the implmentation
namespace gpu {    
    // creates a (m+2) * n array that represents a m*n map of the game
    // with 0 borders on the left and right side
    bool* create_world(int m, int n){
            m = m+2;
            bool * world;// grid to hold randomly generated world
            world = new bool[m*n]();
            srand(time(0));
            //assigning random values to world except for borders
            for(int i = 0; i < n ; i++){
                for(int j = 1; j< m-1; j++){
                    world[(i*m)+j] = rand()%2;
                }
            }
            return world;
        }
    /** 
     *splits a game map in to top and bottom, world itself does not change
     *the report explains this better with graphics, we call this before calling
     *the kernel
     */
     void split(bool **world,bool **top, bool **bot,int m, int n){
        m = m+2;
        int count = m;
        bool *t = *top;
        bool *w = *world;
        bool *b = *bot;
        for(int i = 0; i< m*n; i++){
            t[count]  = w[i];
            count++;
        }
        count = 0;
        for(int i = m; i< m*n; i++){
            b[count] = w[i];
            count++;
        }
    }
    /**
     * kernel, it uses the 3 arrays that represents the map
     * to determine the next iteration of the map and put it in result
     * each thread is responsible for 1 cell
     * there was thought to use shared memory for top core and bot but we never got to it
     */
    __global__
        void next_turn(bool *top, bool *core, bool* bot, bool *result,int m, int n){
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            int living = 0;
            bool current = core[i];
            //this is for not going out of bound of the array
            if(i < m*n){
                //this is for not checking the bouding 0s
                if(i%m != 0 &&(i+1)%m!=0){
                    living = top[i-1] + top[i] + top[i+1] +
                             core[i-1]+         core[i+1] +
                             bot[i-1] + bot[i] + bot[i+1];
                    //we checked there is no problem figuring out the 
                    //number of living cells
                    //printf("%d living: %d\n",i,living);

                    //the condition of determining the result
                    if(living == 3 || (current&& living ==2)){
                        result[i] = true;
                    }else{
                        result[i] = false;
                    }
                }
                //we know the results are here
                //printf("%d",result[i]);
            }
        }

    /**
     * the function that handles allcating device memory and calling setting 
     * everything up for the kernel
     * the main probem encountered was not being able to copy the results back
     * from device to host
     */
    auto bench_mark(void(*func)(bool *,bool *,bool *,bool *,int,int),bool *world, int m, int n,int it){
        
            int size = (n*(m+2))*sizeof(bool);
            bool *top, *bot;
            bool *d_top, *d_core, *d_bot, *d_result;

            //allcating device memories
            hipMalloc((void**)&d_top,size);
            hipMalloc((void **)&d_core,size);
            hipMalloc((void **)&d_bot,size);
            hipMalloc((void **)&d_result,size);
            // an attempt to use unified memory to store results
            // actually able to do so but introduced some other problems
            //cudaMallocManaged((void**)&d_result,size);
            
            //top = (bool *)malloc(size);
            //bot = (bool *)malloc(size);
            //this was used over malloc because we need them to be initialized
            //before sending them to split()
            top = new bool[size]();
            bot = new bool[size]();
            
            //start the timer here so time spend allocating memory would not be counted
            auto const start_time = std::chrono::steady_clock::now();

            //the for loop is to keep repeating the function for the number of iterations
            //it is mostly meaningless since we are unable to get the result back to host
            //a better solution might be to create a global function that hadle this and make
            //so we don't need to have too many back and forth between host and device memory
            for (int i=0; i< it; i++){
                gpu::split(&world,&top,&bot,m,n);

                //copy the world to device memory
                hipMemcpy(d_top, top, size, hipMemcpyHostToDevice);
                hipMemcpy(d_core, world, size, hipMemcpyHostToDevice);
                hipMemcpy(d_bot, bot, size, hipMemcpyHostToDevice);

                //Launching kernel
                gpu::next_turn<<<ceil((m+2)*n/1024),1024>>>(d_top,d_core,d_bot,d_result,m+2,n);
                //we need to wait for the device to finish working before copying the result back otherwise 
                //we will copy the memory before we have done anyhing on the kernel
                hipDeviceSynchronize();

                //we copy the d_result to world so it udates the world for next iteration
                //this does not work for some reason
                //I tested it on other program and it worked accordingly
                //right now the world does not change at all
                hipMemcpy(world, &d_result,size,hipMemcpyDeviceToHost);
                
            }
            auto const end_time = std::chrono::steady_clock::now();
            // enableing these leads to many error
            // not sure why 
            // free(top);free(bot);
            // cudaFree(d_core);cudaFree(d_top);cudaFree(d_bot);cudaFree(d_result);
            return(std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time ).count());
            //cout<< std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time ).count() << " micro seconds\n";
            }
        
        
       

   



}
//not used
namespace cpu{
/**takes the number of living cell and the current bit
 * returns an boolean
 */
    bool dead_or_alive(int living, bool current){
        bool newcell = current;
        //std::cout<< current <<" ";
        if((current == 1 && (living == 3 || living == 2)) || (current == 0 && living == 3)){
            newcell = 1;
        }
        else {
        newcell = 0;
        }
        
        return newcell;
    }
    //creates a matrix of m by n and loading cells with 0|1 and surround it with a border of 0
    bool * create_world(int m, int n)
    {
        bool* world;
        //creating the array so that it is loaded with 0s to avoid if statments. 
        world = new bool[m*n]();
        
        srand(time(0));
        for(int i = 0; i < m ; i++){
            //only setting the inside box to randomized values of 1 or 0
            for(int j = 1; j< n-1; j++){
                world[i*m+j] = rand()%2;
            }
        }
        return world;
    }

    //takes a 2d int matrix and dimentions and print it
    void print_matrix(bool* world,int m, int n){
        for (int i = 0; i < m; i++){
                cout<<"| ";
                for(int j =0; j < n; j++){
                    cout<<world[i*m+j]<<" ";
                }
                cout<<"|\n";
            }
    }

    //return the time it take to run next turn for it number of iterations
    auto bench_mark(bool*(*func)(bool*,int,int),bool* world, int m, int n,int it){
    auto const start_time = std::chrono::steady_clock::now();
    for(int i = 0; i < it; i++){


        world = func(world,m,n);
    }
    auto const end_time = std::chrono::steady_clock::now();
    return(std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time ).count());
    //cout<< std::chrono::duration_cast<std::chrono::microseconds>( end_time - start_time ).count() << " micro seconds\n";
    }

    /**
    * takes a 2d int matrix,and the dimentions
    * goes through each cell
    * determine the next frame of the matrix 
    */
    bool * next_turn(bool* world,int m,int n)
    {
        bool* new_world;
        new_world= new bool[m*n]();
        //#pragma omp parallel for
        for(int i = 1;i < m-1 ;i++){ 
            for(int j = 0; j < n; j++){
                int living = world[(i-1)*m+(j-1)] + world[(i-1)*m+j] + world[(i-1)*m+(j+1)] +
                            world[i*m+(j+1)] + world[i*m+(j-1)] + 
                            world[(i+1)*m+(j-1)] + world[(i+1)*m+j] + world[(i+1)*m+(j+1)];
                new_world[i*m+j] = dead_or_alive(living, world[i*m+j]);
            }
        }
        //free(world); this is slower
        delete[] world;
        return new_world;
    }
}
int main(int argc, char *argv[]){
    if(argc <5 || argc> 6){
        cout<<"Usage: ./gol [width] [height] [iterations] [number of tests][number of threads](if not decleared use all)]";
        return 1;
    }
    //set size larger so we can add border of 0 surrounding the map
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int num_tests = atoi(argv[4]);
    // if(argc==6){
    //     omp_set_num_threads(atoi(argv[5]));
    // }
    //Checking data that was entered making sure it is INT
    if((m == 0) || (n == 0) || (iterations == 0) || (num_tests == 0)){
        cout<< "must enter INT values";
        return 0;
    }
    int time = 0;
    vector<bool*> test_cases;
    //create test cases into a vector
    for (int i = 0; i < num_tests; i++){
        test_cases.push_back(gpu::create_world(m,n));
    }
    // for(int i = 0;i<n;i++){
    //     for(int j =0;j<m+2;j++){
    //         cout<<test_cases[0][i*(m+2)+j];
    //     }
    // cout<< endl;
    // }
    // cout<<"------------\n";
    //run each case
    for(vector<bool*>::iterator i = test_cases.begin(); i != test_cases.end(); ++i){
        time +=gpu::bench_mark(gpu::next_turn,*i,m,n,iterations);
    }
    //print out the average
    cout<< "ran "<< num_tests << " random games of "<< n << " by "<< m << " for "<< iterations<< " iterations, average time is: "<< time/num_tests<<" us"<<endl;
    return 0;
}