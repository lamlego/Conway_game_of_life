
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <time.h> 
using namespace std;

void print_matrix(int** world,int m, int n){
    for (int i = 0; i < m; i++){
            cout<<"| ";
            for(int j =0; j < n; j++){
                cout<<world[i][j]<<" ";
            }
            cout<<"|\n";
        }
    cout<<"\n";
}

int ** create_empty_world(int m, int n){
    int **world = 0;
    world = new int*[m];
    for(int i = 0; i < m; i++){
        world[i] = new int[n];
        for(int j = 0; j < n; j++){
            world[i][j] = 0;
        }
    }

    return world;
}

//just a normal CPU function here. making a world filled with values of 0 or 1 
int ** create_world(int m, int n){
    int **world = 0;
    world = new int*[m];
    int value;
    srand(time(0));
    for (int i = 0; i < m; i ++){
        world[i] = new int[n];
        for(int j =  0 ; j < n; j ++){
            //setting up padding
            if(i == 0 || j == 0 || i == m -2 || j == n -2){
                world[i][j] = 0;
            }else{
                //giving the world[m][n] a value between 1 and 0
                value = rand() % 2; 
                world[i][j] = value;
            }
        }
    }
    return world; 
}


/*
  this will take the world, the new world, the hight and width of the worlds
  We will then go through every value, sum the value of it's 8 neighbours 
  and give a result of 1 or 0 for each cell
  we will do this for the number of turns that we have specified in the command line arguments. 
*/
__global__
void next_turn(int **world, int **new_world, int m, int n ){
    //getting the index that we are currently in 
    int const index_x = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(index_x < m + 1 && index_x != 0){
        for(int index_y = 1; index_y < n - 1 ; index_y++){
            
            int living = world[index_x-1][index_y-1] + world[index_x-1][index_y] + world[index_x-1][index_y+1] +
                        world[index_x][index_y+1] + world[index_x+1][index_y-1] + world[index_x+1][index_y] +
                        world[index_x+1][index_y+1] + world[index_x][index_y-1];

            int current = world[index_x][index_x];
            int new_cell = 0;
            if((current == 1 && (living == 3 || living == 2)) || current == 0 && living == 3){
                new_cell = 1;
            }
            new_world[index_x][index_y] = new_cell;
        }
    }
}

//set up numblocks to be a calculation of how many are needed
int const numblocks = 1;
int const blocksize = 1;
int main(int argc, char *argv[]){
    //verifying the number of arguments
    if(argc <5 || argc> 6){
        cout<<"Usage: ./gol [width] [height] [iterations] [number of tests]";
        return 1;
    }

    //gathering data from the arguments
    int m = atoi(argv[1]);
    m = m + 2;
    int n = atoi(argv[2]);
    n = n + 2;
    int iterations = atoi(argv[3]);
    int num_tests = atoi(argv[4]);

    int size = sizeof(int) * m * n; 

    //verifying the arguments are able to be converted to int values
    if((m == 0) || (n == 0) || (iterations == 0) || (num_tests == 0)){
        cout<< "must enter INT values";
        return 0;
    }

    //create a world and a new world
    int **world = create_world(m, n);
    int **new_world = create_empty_world(m, n);

    print_matrix(world, m, n); 

    cout<< "about to copy stuff over";
    //create a matrix of ints for the device
    int **dev_world;
    int **dev_new_world;
    hipMalloc((void **)&dev_world, size);
    hipMalloc((void **)&dev_new_world, size);
    if(dev_world == NULL) {std::cerr << "not able to alloc memory on device" << std::endl;}
    if(dev_new_world == NULL) {std::cerr << "not able to alloc memory on device" << std::endl;}

    //copy values from host to device
    hipMemcpy(dev_world, world, size, hipMemcpyHostToDevice);
    //cudaMemcpy(dev_new_world, new_world, size, cudaMemcpyHostToDevice);
    //no output yet, don't need to copy it over

    //set up timing here, no need to take into account how long it takes to copy stuffs over. 

    //do next_turn on GPU
    for(int i = 0; i < iterations; i ++){
        next_turn<<< numblocks, blocksize >>>(dev_world, dev_new_world, m, n);
    }

    //copy result back to host
    hipMemcpy(new_world, dev_new_world, size, hipMemcpyDeviceToHost);

    print_matrix(new_world, m, n);
    //free space that we created
    for(int i = 0; i < m ; i++){
        free(world[i]);
        free(new_world[i]);
    }
    free(world);
    free(new_world);
    hipFree(dev_new_world);
    hipFree(dev_world);
}