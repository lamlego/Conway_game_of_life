#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <time.h> 
using namespace std;

void print_matrix(int** world,int m, int n){
    for (int i = 0; i < m; i++){
            cout<<"| ";
            for(int j =0; j < n; j++){
                cout<<world[i][j]<<" ";
            }
            cout<<"|\n";
        }
}

int ** create_empty_world(int m, int n){
    int **world = new int*[m];
    for(int i = 0; i < m; i++){
        world[i] = new int[n];
        for(int j = 0; j < n; j++){
            world[i][j] = 0;
        }
    }

    return world;
}

//just a normal CPU function here. making a world filled with values of 0 or 1 
int ** create_world(int m, int n){
    int **world = new int[m];
    int value;
    for (int i = 0; i < m; i ++){
        world[i] = new int[n];
        for(int j =  0 ; j < n; j ++){
            //setting up padding
            if(i == 0 || j == 0 || i == m || j == n){
                world[i][j] = 0;
            }else{
                srand(time(NULL));
                //giving the world[m][n] a value between 1 and 0
                value = rand() % 2; 
                world[i][j] = value;
            }
        }
    }
    return world; 
}

int dead_or_alive(int living, int current){
    int newcell = current;
    //std::cout<< current <<" ";
    if((current == 1 && (living == 3 || living == 2)) || (current == 0 && living == 3)){
        newcell = 1;
    }
    else {
       newcell = 0;
    }
    
    return newcell;
}

/*
  this will take the world, the new world, the hight and width of the worlds
  We will then go through every value, sum the value of it's 8 neighbours 
  and give a result of 1 or 0 for each cell
  we will do this for the number of turns that we have specified in the command line arguments. 
*/
__global__
void next_turn(int **world, int **new_world, int m, int n ){
    //getting the index that we are currently in 
    int const index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int const index_y = threadIdx.y + blockIdx.y * blockDim.y;
    if(index_x < m){
        if(index_y < n){
            int living = world[index_x-1][index_y-1] + world[index_x-1][index_y] + world[index_x-1][index_y+1] +
                        world[index_x][index_y+1] + world[index_x+1][index_y-1] + world[index_x+1][index_y] +
                        world[index_x+1][index_y+1] + world[index_x][index_y-1];
            new_world[index_x][index_y] = dead_or_alive(living, world[index_x][index_y]);
        }
    }
}

#define NUM_BLOCKS 1
#define NUM_THREADS_PER_BLOCK 1;
int main(int argc, char *argv[]){
    //verifying the number of arguments
    if(argc <5 || argc> 6){
        cout<<"Usage: ./gol [width] [height] [iterations] [number of tests]";
        return 1;
    }

    //gathering data from the arguments
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int num_tests = atoi(argv[4]);

    int size = sizeof(int) * m * n; 

    //verifying the arguments are able to be converted to int values
    if((m == 0) || (n == 0) || (iterations == 0) || (num_tests == 0)){
        cout<< "must enter INT values";
        return 0;
    }

    //create a world and a new world
    int **world = create_world(m + 2, n + 2);
    int **new_world = create_empty_world(m + 2, n + 2);

    print_matrix(world, m, n); 

    //create a matrix of ints for the device
    int **dev_world;
    int **dev_new_world;
    hipMalloc((void **)&dev_world, size);
    hipMalloc((void **)&dev_new_world, size);
    if(dev_world == NULL) {std::cerr << "not able to alloc memory on device" << std::endl;}
    if(dev_new_world == NULL) {std::cerr << "not able to alloc memory on device" << std::endl;}

    //copy values from host to device
    hipMemcpy(dev_world, world, size, hipMemcpyHostToDevice);
    //hipMemcpy(dev_new_world, new_world, size, hipMemcpyHostToDevice);
    //no output yet, don't need to copy it over

    //do next_turn on GPU
    for(int i = 0; i < iterations; i ++){
        next_turn<<< NUM_BLOCKS, NUM_BLOCKS, NUM_THREADS_PER_BLOCK >>>(dev_world, dev_new_world, m, n);
    }

    //copy result back to host
    hipMemcpy(new_world, dev_new_world, size, hipMemcpyDeviceToHost);

    print_matrix(new_world, m, n);
    //free space that we created
    hipFree(dev_new_world);
    hipFree(dev_world);
}