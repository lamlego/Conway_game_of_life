#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <time.h> 
using namespace std;

int ** create_empty_world(int m, int n){
    int world = new int[m];
    for(int i = 0; i < m; i++){
        world[i] = new int[n];
        for(int j; j < n; j++){
            world[i][j] = 0;
        }
    }

    return world;
}

//just a normal CPU function here. making a world filled with values of 0 or 1 
int ** create_world(int m, int n){
    int world = new int[m];
    int value;
    for (int i = 0; i < m; i ++){
        world[i] = new int[n];
        for(int j =  0 ; j < n; j ++){
            srand(time(NULL));
            //giving the world[m][n] a value between 1 and 0
            value = rand() % 2; 
            world[i][j] = value;
        }
    }
    return world; 
}

__global__
void next_turn(int **world, int **new_world, int m, int n ){
    //getting the index that we are currently in 
    int const index = threadIdx.x + blockIdx.x * blockDim.x;
    
}

#define NUM_BLOCKS 1
#define NUM_THREADS_PER_BLOCK 1;
int main(int argc, char *argv[]){
    //verifying the number of arguments
    if(argc <5 || argc> 6){
        cout<<"Usage: ./gol [width] [height] [iterations] [number of tests]";
        return 1;
    }

    //gathering data from the arguments
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int iterations = atoi(argv[3]);
    int num_tests = atoi(argv[4]);

    int size = sizeof(int) * m * n; 

    //verifying the arguments are able to be converted to int values
    if((m == 0) || (n == 0) || (iterations == 0) || (num_tests == 0)){
        cout<< "must enter INT values";
        return 0;
    }

    //create a world and a new world
    int **world = create_world(m, n);
    int **new_world = create_empty_world(m, n);

    //create a matrix of ints for the device
    int **dev_world;
    int **dev_new_world;
    hipMalloc((void **)&dev_world, size);
    hipMalloc((void **)&dev_new_world, size);

    //copy values from host to device
    hipMemcpy(dev_world, world, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_new_world, new_world, size, hipMemcpyHostToDevice);

    //do next_turn on GPU

    //free space that we created
    hipFree(dev_new_world);
    hipFree(dev_world);
}