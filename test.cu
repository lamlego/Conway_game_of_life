
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sstream>
#include <chrono>	 // timing libraries
#include <vector>
#include <algorithm>
#include <omp.h>
using namespace std;

typedef struct {
    bool *core;
    bool *top;
    bool *bot;
} game_map;
//generate a game map with side borders but not above or below
bool* create_world(int m, int n){
    m = m+2;
    bool * world;//temperory grid to hold randomly generated world
    world = new bool[m*n]();
    srand(time(0));
    //assigning random values to world except for borders
    for(int i = 0; i < n ; i++){
        for(int j = 1; j< m-1; j++){
            world[(i*m)+j] = rand()%2;
        }
    }
    return world;
}
//takes a world and split it into 3 parts
void primal_split(bool **world,bool **top, bool **bot,int m, int n){
    m = m+2;
    int count = m;
    bool *t = *top;
    bool *w = *world;
    bool *b = *bot;
    for(int i = 0; i< m*n; i++){
        t[count] = w[i];
        //cout<<w[i];
        count++;
    }
    count = 0;
    for(int i = m; i< m*n; i++){

        b[count] = w[i];
        count++;
    }
}

__global__
void flip(bool *top, bool *core, bool *bot, int *res){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("hi %d\n",core[i]);
    res[i] = top[i]+core[i]+bot[i];
    

}

int main(int argc, char *argv[]){
    //cout<<sizeof(game_map)<<"\n";
    bool *map = create_world(10,10);
    bool *top,*bot;
    bool *d_top, *d_core, *d_bot;
    int *res, *d_res;
    res = (int *)malloc(120*sizeof(int));
    //top = (bool *)malloc(120*sizeof(bool));
    top = new bool[120]();
    bot = new bool[120]();
    //bot = (bool *)malloc(120*sizeof(bool));
    primal_split(&map,&top,&bot,10,10);

    hipMalloc((void **)&d_top,120*sizeof(bool));
    hipMalloc((void **)&d_core,120*sizeof(bool));
    hipMalloc((void **)&d_bot,120*sizeof(bool));
    hipMalloc((void **)&d_res,120*sizeof(int));
    
    hipMemcpy(d_top,top,120*sizeof(bool),hipMemcpyHostToDevice);
    hipMemcpy(d_core,map,120*sizeof(bool),hipMemcpyHostToDevice);
    hipMemcpy(d_bot,bot,120*sizeof(bool),hipMemcpyHostToDevice);
    
    flip<<<10,12>>>(d_top,d_core,d_bot,d_res);

    hipMemcpy(res,&d_res,120*sizeof(int),hipMemcpyDeviceToHost);
    // for(int i = 0;i<10;i++){
    //     for(int j =0;j<12;j++){
    //         cout<<top[i*12+j];
    //     }
    //     cout<< endl;
    // }
    // cout<<"-------\n";
    // for(int i = 0;i<10;i++){
    //     for(int j =0;j<12;j++){
    //         cout<<map[i*12+j];
    //     }
    //     cout<< endl;
    // }
    // cout<<"-------\n";
    // for(int i = 0;i<10;i++){
    //     for(int j =0;j<12;j++){
    //         cout<<bot[i*12+j];
    //     }
    //     cout<< endl;
    // }
    // cout<<"-------\n";
    //cout<<sizeof(map.core);
    for(int i = 0;i<10;i++){
        for(int j =0;j<12;j++){
            cout<<res[i*12+j];
        }
        cout<< endl;
    }
}