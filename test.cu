
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sstream>
#include <chrono>	 // timing libraries
#include <vector>
#include <algorithm>
#include <omp.h>
using namespace std;

typedef struct {
    bool *core;
    bool *top;
    bool *bot;
} game_map;
//generate a game map with side borders but not above or below
bool* create_world(int m, int n){
    m = m+2;
    bool * world;//temperory grid to hold randomly generated world
    world = new bool[m*n]();
    srand(time(0));
    //assigning random values to world except for borders
    for(int i = 0; i < n ; i++){
        for(int j = 1; j< m-1; j++){
            world[(i*m)+j] = rand()%2;
        }
    }
    return world;
}
//takes a world and split it into 3 parts
// void primal_split(bool **world,bool **top, bool **bot,int m, int n){
//     m = m+2;
//     int count = m;
//     bool *t = *top;
//     bool *w = *world;
//     bool *b = *bot;
//     for(int i = 0; i< m*n; i++){
//         t[count] = w[i];
//         //cout<<w[i];
//         count++;
//     }
//     count = 0;
//     for(int i = m; i< m*n; i++){

//         b[count] = w[i];
//         count++;
//     }
// }

__global__
void flip(bool *top, bool *core, bool *bot, int *res,int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n){
        printf("hi %d\n",core[i]);
            res[i] = top[i]+core[i]+bot[i];
            
    }
    

}

__global__
void add(int *a,int *b,int *c,int n){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    c[i]= a[i] +b[i];

}

int main(int argc, char *argv[]){
    int size = 10* sizeof(int);
    int *a, *b, *c;
    int *d_a, *d_b, *d_c; 
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);   
    for(int i=0;i<10;i++){
        a[i]=i;
        b[i]=i;
    }
    cout<<a[1];
    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    add<<<1,32>>>(d_a,d_b,d_c,10);
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

    for(int i=0;i<10;i++){
        cout<<c[i]<<" ";
    }
    //cout<<sizeof(game_map)<<"\n";
    // bool *top,*bot;
    // bool *d_top, *d_core, *d_bot;
    // int *res, *d_res;
    // res = (int *)malloc(120*sizeof(int));
    // //top = (bool *)malloc(120*sizeof(bool));
    // top = new bool[120]();
    // bot = new bool[120]();
    // //bot = (bool *)malloc(120*sizeof(bool));
    // primal_split(&map,&top,&bot,10,10);

    // cudaMalloc((void **)&d_top,120*sizeof(bool));
    // cudaMalloc((void **)&d_core,120*sizeof(bool));
    // cudaMalloc((void **)&d_bot,120*sizeof(bool));
    // cudaMalloc((void **)&d_res,120*sizeof(int));
    
    // cudaMemcpy(d_top,top,120*sizeof(bool),cudaMemcpyHostToDevice);
    // cudaMemcpy(d_core,map,120*sizeof(bool),cudaMemcpyHostToDevice);
    // cudaMemcpy(d_bot,bot,120*sizeof(bool),cudaMemcpyHostToDevice);
    
    // flip<<<10,32>>>(d_top,d_core,d_bot,d_res,120);

    // cudaMemcpy(res,&d_res,120*sizeof(int),cudaMemcpyDeviceToHost);
    // for(int i = 0;i<10;i++){
    //     for(int j =0;j<12;j++){
    //         cout<<top[i*12+j];
    //     }
    //     cout<< endl;
    // }
    // cout<<"-------\n";
    // for(int i = 0;i<10;i++){
    //     for(int j =0;j<12;j++){
    //         cout<<map[i*12+j];
    //     }
    //     cout<< endl;
    // }
    // cout<<"-------\n";
    // for(int i = 0;i<10;i++){
    //     for(int j =0;j<12;j++){
    //         cout<<bot[i*12+j];
    //     }
    //     cout<< endl;
    // }
    // cout<<"-------\n";
    //cout<<sizeof(map.core);
    // for(int i = 0;i<10;i++){
    //     for(int j =0;j<12;j++){
    //         cout<<res[i*12+j];
    //     }
    //     cout<< endl;
    // }
}