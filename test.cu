
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sstream>
#include <chrono>	 // timing libraries
#include <vector>
#include <algorithm>
#include <omp.h>
using namespace std;

// typedef struct {
//     bool *core;
//     bool *top;
//     bool *bot;
// } game_map;
// //generate a game map with side borders but not above or below
// bool* create_world(int m, int n){
//     m = m+2;
//     bool * world;//temperory grid to hold randomly generated world
//     world = new bool[m*n]();
//     srand(time(0));
//     //assigning random values to world except for borders
//     for(int i = 0; i < n ; i++){
//         for(int j = 1; j< m-1; j++){
//             world[(i*m)+j] = rand()%2;
//         }
//     }
//     return world;
// }
//takes a world and split it into 3 parts
// void primal_split(bool **world,bool **top, bool **bot,int m, int n){
//     m = m+2;
//     int count = m;
//     bool *t = *top;
//     bool *w = *world;
//     bool *b = *bot;
//     for(int i = 0; i< m*n; i++){
//         t[count] = w[i];
//         //cout<<w[i];
//         count++;
//     }
//     count = 0;
//     for(int i = m; i< m*n; i++){

//         b[count] = w[i];
//         count++;
//     }
// }

// __global__
// void flip(bool *top, bool *core, bool *bot, int *res,int n){
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if(i<n){
//         printf("hi %d\n",core[i]);
//             res[i] = top[i]+core[i]+bot[i];
            
//     }
    

// }

__global__
void add(int *a,int *b,int *c,int n){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    c[i]= a[i] +b[i];

}

int main(int argc, char *argv[]){
    int size = 100000* sizeof(int);
    int *a, *b, *c;
    int *d_a, *d_b, *d_c; 
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);   
    for(int i=0;i<100000;i++){
        a[i]=i;
        b[i]=i;
    }
    cout<<a[0]<<"first element in a\n";
    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    cout<<"before calling kernel\n";
    add<<<10,64>>>(d_a,d_b,d_c,100000);
    cout<<"after calling kernel\n";
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
    cout<<"first 100 elements in sum vector c\n";
    for(int i=0;i<100;i++){
        cout<<c[i]<<" ";
    }
 
}